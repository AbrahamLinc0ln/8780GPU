#include "hip/hip_runtime.h"
#include "im2Gray.h"

#define BLOCK 32



/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__
void im2Gray_share(uchar4 *d_in, unsigned char *d_gray, int numRows, int numCols, int TILE_WIDTH){
	
}

__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_gray, int numRows, int numCols){

 /*
   Your kernel here: Make sure to check for boundary conditions
  */
		
  int x = threadIdx.x + blockIdx.x*blockDim.x;
  int y = threadIdx.y + blockIdx.y*blockDim.y;
  if (y < numRows && x < numCols){
	int pointIndex = y*numRows + x;
	uchar4 imagePoint = d_in[pointIndex];
	d_gray[pointIndex] = .299f*imagePoint.x + .587f*imagePoint.y  + .114f*imagePoint.z;
  }
}




void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols){
    // configure launch params here 
    int x_thread = 32;
    int y_thread = 32;
    int grid_x = numCols/x_thread;
    int grid_y = numRows/y_thread; 
    dim3 block(x_thread,y_thread,1);
    dim3 grid(ceil(grid_x),ceil(grid_y), 1);

    im2Gray<<<grid,block>>>(d_in, d_grey, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    
}





